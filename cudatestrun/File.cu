#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "./header.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <chrono>
using namespace std::chrono;

#define N 100000000
#define N 100000000
#define NUMTHREADS 512

//#define N 10000
//#define NUMTHREADS 100


__device__ bool isprime(int n)
{
    if (n <= 1)
        return false;
    int s = sqrt((float)n);

    for (int i = 2; i <= s; ++i)
    {
        if (n % i == 0)
            return false;
    }
    return true;
}

__global__ void kernel_calc_primes_dev(int* a,int* primecount)
{
    __shared__ float cache[NUMTHREADS];
    int tid = threadIdx.x;
    int count = 0;
    while (tid < N)
    {
        if (isprime(a[tid]))
            ++count;
        tid += NUMTHREADS;
    }

    cache[threadIdx.x] = count;
    __syncthreads();

    if (threadIdx.x == 0)
    {
        for (int i = 0; i < NUMTHREADS;++i)
        {
            *primecount += cache[i];
        }
    }
}

void kernel_calc_primes_host()
{
    int c;
    int *dev_c;
    int* a = new int[N];
    for (int i = 0; i < N; ++i)
    {
        a[i] = i + 1;
    }
    int* dev_a;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMalloc((void**)&dev_c, sizeof(int));
    hipMalloc((void**)&dev_a, sizeof(int) * N);

    hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyKind::hipMemcpyHostToDevice);

    kernel_calc_primes_dev <<< 1, NUMTHREADS >>> (dev_a,dev_c);

    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_c);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("%d primes found between [%d] and [%d] in %d seconds.", c,1,N, (int)elapsedTime/1000);

    delete[] a;

}

